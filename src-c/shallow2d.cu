#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <math.h>

#define BLOCKS 512
#define THREADS_PER_BLOCK 1024 

//ldoc on
/**
 * ## Implementation
 *
 * The actually work of computing the fluxes and speeds is done
 * by local (`static`) helper functions that take as arguments
 * pointers to all the individual fields.  This is helpful to the
 * compilers, since by specifying the `restrict` keyword, we are
 * promising that we will not access the field data through the
 * wrong pointer.  This lets the compiler do a better job with
 * vectorization.
 */

extern "C" {
#include "shallow2d.h"
}

static const float g = 9.8;

static
void shallow2dv_flux(float* __restrict__ fh,
                     float* __restrict__ fhu,
                     float* __restrict__ fhv,
                     float* __restrict__ gh,
                     float* __restrict__ ghu,
                     float* __restrict__ ghv,
                     const float* __restrict__ h,
                     const float* __restrict__ hu,
                     const float* __restrict__ hv,
                     float g,
                     int ncell)
{
    memcpy(fh, hu, ncell * sizeof(float));
    memcpy(gh, hv, ncell * sizeof(float));
    for (int i = 0; i < ncell; ++i) {
        float hi = h[i], hui = hu[i], hvi = hv[i];
        float inv_h = 1/hi;
        fhu[i] = hui*hui*inv_h + (0.5f*g)*hi*hi;
        fhv[i] = hui*hvi*inv_h;
        ghu[i] = hui*hvi*inv_h;
        ghv[i] = hvi*hvi*inv_h + (0.5f*g)*hi*hi;
    }
}


static
void shallow2dv_speed(float* __restrict__ cxy,
                      const float* __restrict__ h,
                      const float* __restrict__ hu,
                      const float* __restrict__ hv,
                      float g,
                      int ncell)
{
    float cx = cxy[0];
    float cy = cxy[1];
    for (int i = 0; i < ncell; ++i) {
        float hi = h[i];
        float inv_hi = 1.0f/h[i];
        float root_gh = sqrtf(g * hi);
        float cxi = fabsf(hu[i] * inv_hi) + root_gh;
        float cyi = fabsf(hv[i] * inv_hi) + root_gh;
        if (cx < cxi) cx = cxi;
        if (cy < cyi) cy = cyi;
    }
    cxy[0] = cx;
    cxy[1] = cy;
}

void shallow2d_flux(float* FU, float* GU, const float* U,
                    int ncell, int field_stride)
{
    shallow2dv_flux(FU, FU+field_stride, FU+2*field_stride,
                    GU, GU+field_stride, GU+2*field_stride,
                    U,  U +field_stride, U +2*field_stride,
                    g, ncell);
}

/*
void shallow2d_speed(float* cxy, const float* U,
                     int ncell, int field_stride)
{
    shallow2dv_speed(cxy, U, U+field_stride, U+2*field_stride, g, ncell);
}
*/
/**
 *  Compute the maximum of 2 single-precision floating point values using an atomic operation
 *
 * @param[in]	address	The address of the reference value which might get updated with the minimum
 * @param[in]	value	The value that is compared to the reference in order to determine the minimum
 */

__device__
void AtomicMax(float * const address, const float value)
{
    if (* address >= value) { return; }

    int * const address_as_i = (int *)address;
    int old = * address_as_i, assumed;

    do {
        assumed = old;
        if (__int_as_float(assumed) >= value) { break; }

        old = atomicCAS(address_as_i, assumed, __float_as_int(value));
        } while (assumed != old);
}

__global__
void cuda_speed (float* cx,
                 float* cy,
                 const float* h,
                 const float* hu,
                 const float* hv)
{
  __shared__ float tempx[THREADS_PER_BLOCK];
  __shared__ float tempy[THREADS_PER_BLOCK];
  static const float g = 9.8;
  int index = threadIdx.x;

  float hi = h[index];
  float inv_hi = 1.0f/h[index];
  float root_gh = sqrtf(g * hi);
  tempx[index] = fabsf(hu[index] * inv_hi) + root_gh;
  tempy[index] = fabsf(hv[index] * inv_hi) + root_gh;

  __syncthreads();

  if ( 0 == threadIdx.x ) {
    float x, y = 0.0;
    for (int i = 0 ; i < THREADS_PER_BLOCK; i++)  {
      x = fmaxf(x, tempx[i]);
      y = fmaxf(y, tempy[i]);
    }
    // *cx = fmaxf(*cx,x);
    // *cy = fmaxf(*cy,y);
    AtomicMax(cx, x);
    AtomicMax(cy, y);
  }
}

void shallow2d_speed(float* cxy, const float* U,
                     int ncell, int field_stride)
{
    int size = 3 * ncell * sizeof(float);
    float *cuda_U, *cx, *cy;
    
    hipMalloc( (void**)&cuda_U, size );
    hipMalloc( (void**)&cx, sizeof(float) );
    hipMalloc( (void**)&cy, sizeof(float) );

    hipMemcpy( cx, &cxy[0], sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( cy, &cxy[1], sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( cuda_U, U, size, hipMemcpyHostToDevice );

    //shallow2dv_speed(cxy, U, U+field_stride, U+2*field_stride, g, ncell);
    cuda_speed<<<BLOCKS, THREADS_PER_BLOCK>>>(cx, cy, cuda_U, cuda_U+field_stride, cuda_U+2*field_stride);
    hipMemcpy(&cxy[0], cx, sizeof(float),  hipMemcpyDeviceToHost);
    hipMemcpy(&cxy[1], cy, sizeof(float),  hipMemcpyDeviceToHost);

    hipFree(cuda_U);
    hipFree(cx);
    hipFree(cy);
}
